#include "hip/hip_runtime.h"
/*
Host: CPU
Device: GPU
    __global__ - Runs on the GPU, called from the CPU or the GPU*. Executed with <<<dim3>>> arguments.
    __device__ - Runs on the GPU, called from the GPU. Can be used with variabiles too.
    __host__ - Runs on the CPU, called from the CPU.
     __global__ functions can be called from other __global__ functions starting compute capability 3.5.
*/
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#include <iostream>
#include <sstream>
#include <cmath>
#include <random>
#include <fstream>
#include <array>
#include <string>

struct Atom {
    std::array<double, 3> positions;
    std::array<double, 3> velocities;
    std::array<double, 3> accelerations;
    std::array<double, 3> oldAccelerations;

    Atom(double x, double y, double z) {
        positions[0] = x;
        positions[1] = y;
        positions[2] = z;
        accelerations = {0, 0, 0};
        oldAccelerations = {0, 0, 0};
    }
};

const double Kb = 1.38064582 * std::pow(10, -23); // J / K
const double Na = 6.022 * std::pow(10, 23); // Atoms per mole

const int numTimeSteps = 5000; // Parameters to change for simulation
const double dt_star= .001;

const int N = 4000; // Number of atoms in simulation
const double SIGMA = 3.405; // Angstroms
const double EPSILON = 1.6540 * std::pow(10, -21); // Joules
const double EPS_STAR = EPSILON / Kb; // ~ 119.8 K

const double rhostar = .45; // Dimensionless density of gas
const double rho = rhostar / std::pow(SIGMA, 3); // Density of gas
const double L = std::cbrt(N / rho); // Unit cell length
const double rCutoff = SIGMA * 2.5; // Forces are negligible past this distance, not worth calculating
const double rCutoffSquared = rCutoff * rCutoff;
const double tStar = 1.24; // Reduced units of temperature
const double TARGET_TEMP = tStar * EPS_STAR;
// 39.9 is mass of argon in amu, 10 is a conversion between the missing units :)
const double MASS = 39.9 * 10 / Na / Kb; // Kelvin * ps^2 / A^2
const double timeStep = dt_star * std::sqrt(MASS * SIGMA * SIGMA / EPS_STAR); // Convert time step to picoseconds

const double targetCellLength = rCutoff;
const int numCellsPerDirection = std::floor(L / targetCellLength);
const double cellLength = L / numCellsPerDirection; // Side length of each cell

double dot(double x, double y, double z);
void thermostat(thrust::host_vector<Atom> &atomList);
double calcForces(thrust::host_vector<Atom> &atomList, int (&cellInteractionIndexes)[343][14]);
thrust::host_vector<Atom> faceCenteredCell();
std::vector<Atom> simpleCubicCell();
void radialDistribution();



__host__
void writePositions(thrust::host_vector<Atom> &atomList, std::ofstream &positionFile, int i) {
    positionFile << N << "\nTime: " << i << "\n";
    for (int j = 0; j < N; ++j) { // Write positions to xyz file
        positionFile << "A " << atomList[j].positions[0] << " " << atomList[j].positions[1] << " " << atomList[j].positions[2] << "\n";
    }
}

__host__
int calcCellIndex(int x, int y, int z) {
    return x * numCellsPerDirection * numCellsPerDirection + y * numCellsPerDirection + z;
}

__host__
std::array<int, 3> calcCellFromIndex(int index) {
    std::array<int, 3> arr;
    int numCellsYZ = numCellsPerDirection * numCellsPerDirection;
    arr[0] = index / numCellsYZ;
    int remainder = index % numCellsYZ;
    arr[1] = remainder / numCellsPerDirection;
    arr[2] = remainder % numCellsPerDirection;
    return arr;
}

__host__
std::array<int, 3> moveCellInsideBox(int x, int y, int z) {
    std::array<int, 3> cell;
    cell[0] = (x + numCellsPerDirection) % numCellsPerDirection;
    cell[1] = (y + numCellsPerDirection) % numCellsPerDirection;
    cell[2] = (z + numCellsPerDirection) % numCellsPerDirection;
    return cell;
}

__host__
int processCell(int x, int y, int z) {
    std::array<int, 3> shiftedNeighbor = moveCellInsideBox(x, y, z);
    int index = calcCellIndex(shiftedNeighbor[0], shiftedNeighbor[1], shiftedNeighbor[2]);
    return index;
}

__host__
void calcCellInteractions(std::vector<std::vector<int>> &cellInteractionIndexes, int numCellsXYZ) {
    for (int i = 0; i < numCellsXYZ; i++) {
        std::vector<int> arr;
        std::array<int, 3> cell = calcCellFromIndex(i);

        arr.push_back(processCell(cell[0], cell[1], cell[2]));
        arr.push_back(processCell(cell[0], cell[1], cell[2] + 1));
        arr.push_back(processCell(cell[0], cell[1] + 1, cell[2] - 1));
        arr.push_back(processCell(cell[0], cell[1] + 1, cell[2]));
        arr.push_back(processCell(cell[0], cell[1] + 1, cell[2] + 1));

        // Next level above
        arr.push_back(processCell(cell[0] + 1, cell[1] - 1, cell[2] - 1));
        arr.push_back(processCell(cell[0] + 1, cell[1] - 1, cell[2]));
        arr.push_back(processCell(cell[0] + 1, cell[1] - 1, cell[2] + 1));
        arr.push_back(processCell(cell[0] + 1, cell[1], cell[2] - 1));
        arr.push_back(processCell(cell[0] + 1, cell[1], cell[2]));
        arr.push_back(processCell(cell[0] + 1, cell[1], cell[2] + 1));
        arr.push_back(processCell(cell[0] + 1, cell[1] + 1, cell[2] - 1));
        arr.push_back(processCell(cell[0] + 1, cell[1] + 1, cell[2]));
        arr.push_back(processCell(cell[0] + 1, cell[1] + 1, cell[2] + 1));

        cellInteractionIndexes[i] = arr;
    }
}

int main() {
    int numCellsYZ = numCellsPerDirection * numCellsPerDirection;
    int numCellsXYZ = numCellsPerDirection * numCellsYZ;
    std::vector<std::vector<int>> vectorIndicies;

    std::cout << "Cells per direction: " << numCellsPerDirection << std::endl;
    std::cout << "Simulation length: " << L << std::endl;
    std::cout << "Cell length: " << cellLength << std::endl;

    std::ofstream positionFile("out.xyz");
    //std::ofstream debug("debug.dat");
    //debug << "I \t J \t C \t C1 \t R2 \t forceOverR \n";

    calcCellInteractions(vectorIndicies, numCellsXYZ);

    int cellInteractionIndexes[numCellsXYZ][14]; // Interactions indexes in an array to make CUDA happy

    for (int i = 0; i < vectorIndicies.size(); i++) {
        for (int j = 0; j < vectorIndicies[i].size(); j++) {
            cellInteractionIndexes[i][j] = vectorIndicies[i][j];
        }
    }


    // Arrays to hold energy values at each step of the process
    std::vector<double> KE;
    std::vector<double> PE;
    std::vector<double> netE;

    std::random_device rd;
    std::default_random_engine generator(3); // (rd())
    std::uniform_real_distribution<double> distribution(-1.0, 1.0);

    thrust::host_vector<Atom> atomList = faceCenteredCell();

    for (int i = 0; i < N; ++i) { // Randomize velocities
         for (int j = 0; j < 3; ++j) {
             atomList[i].velocities[j] = distribution(generator);
         }
    }
   
    thermostat(atomList); // Make velocities more accurate

    double totalVelSquared;
    double netPotential;

    double count = .01;
    for (int i = 0; i < numTimeSteps; ++i) { // Main loop handles integration and printing to files

        if (i > count * numTimeSteps) { // Percent progress
            std::cout << count * 100 << "% \n";
            count += .01;
        }

        writePositions(atomList, positionFile, i);

        for (int k = 0; k < N; ++k) { // Update positions
            for (int j = 0; j < 3; ++j) {
                atomList[k].positions[j] += atomList[k].velocities[j] * timeStep 
                    + .5 * atomList[k].accelerations[j] * timeStep * timeStep;
                atomList[k].positions[j] += -L * std::floor(atomList[k].positions[j] / L); // Keep atom inside box
                atomList[k].oldAccelerations[j] = atomList[k].accelerations[j];
            }
        }

        netPotential = calcForces(atomList, cellInteractionIndexes); // Update accelerations and return potential of system

        totalVelSquared = 0;
        for (int k = 0; k < N; ++k) { // Update velocities
            for (int j = 0; j < 3; ++j) {
                atomList[k].velocities[j] += .5 * (atomList[k].accelerations[j] + atomList[k].oldAccelerations[j]) * timeStep;
                totalVelSquared += atomList[k].velocities[j] * atomList[k].velocities[j];
            }
        }

        if (i < numTimeSteps / 2 && i % 5 == 0) { // Apply velocity modifications for first half of sample
            thermostat(atomList);
        }

        if (i > numTimeSteps / 2) { // Record energies after half of time has passed
            double netKE = .5 * MASS * totalVelSquared;
            KE.push_back(netKE);
            PE.push_back(netPotential);
            netE.push_back(netPotential + netKE);
        }
    }

    double avgPE = 0; // Average PE array
    for (double i : PE) {
        avgPE += i;
    }
    avgPE /= PE.size();

    double SoLo2 = SIGMA / (L / 2); // Sigma over L over 2
    double Ulrc = (8.0 / 3.0) * M_PI * N * rhostar * EPS_STAR; // Potential sub lrc (long range corrections)
    double temp = 1.0 / 3.0 * std::pow(SoLo2, 9.0);
    double temp1 = std::pow(SoLo2, 3.0);
    Ulrc *= (temp - temp1);
    double PEstar = ((avgPE + Ulrc) / N) / EPS_STAR; // Reduced potential energy

    std::cout << "Reduced potential with long range correction: " << PEstar << std::endl;

    positionFile.close();
    //debug.close();

    // std::cout << "Finding radial distribution \n";
    // radialDistribution(); // Comment out function to reduce runtime

    return 0;
}

__device__
void dotForGPU(double x, double y, double z, double &r2) { // Returns dot product of a vector
    r2 = x * x + y * y + z * z;
}

__host__
double dotForCPU(double x, double y, double z) {
    return x * x + y * y + z * z;
}

__host__
void thermostat(thrust::host_vector<Atom> &atomList) {
    double instantTemp = 0;
    for (int i = 0; i < N; i++) { // Add kinetic energy of each molecule to the temperature
        instantTemp += MASS * dotForCPU(atomList[i].velocities[0], atomList[i].velocities[1], atomList[i].velocities[2]);
    }
    instantTemp /= (3 * N - 3);
    double tempScalar = std::sqrt(TARGET_TEMP / instantTemp);
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < 3; ++j) {
            atomList[i].velocities[j] *= tempScalar; // V = V * lambda
        }
    }
}

__global__
void calcForcesOnCell(int cellI, thrust::device_vector<Atom> &gpuAtom, int (&cellInteractionIndexes)[343][14], thrust::device_vector<thrust::device_vector<int>> &atomsInCells) {
    double distArr[3]; // Record distance between atoms
    double netPotential = 0;
    auto cellArr = atomsInCells[cellI];
    double r2;

    // Scan neighbor cells including the one currently active
    for (int cellJ : cellInteractionIndexes[cellI]) {
        auto neighborCellArr = atomsInCells[cellJ];

        for (int atomi : cellArr) {
            for (int atomj : neighborCellArr) {
                if (atomi < atomj || cellI != cellJ) { // Don't double count atoms (if i > j its already been counted)
                    for (int k = 0; k < 3; k++) {
                        // Apply boundary conditions
                        distArr[k] = gpuAtom[atomi].positions[k] - gpuAtom[atomj].positions[k];
                        distArr[k] -= L * std::round(distArr[k] / L);
                    }
                    dotForGPU(distArr[0], distArr[1], distArr[2], r2); // Dot of distance vector between the two atoms
                    if (r2 < rCutoffSquared) {
                        double s2or2 = SIGMA * SIGMA / r2; // Sigma squared over r squared
                        double sor6 = s2or2 * s2or2 * s2or2; // Sigma over r to the sixth
                        double sor12 = sor6 * sor6; // Sigma over r to the twelfth

                        double forceOverR = 24 * EPS_STAR / r2 * (2 * sor12 - sor6);
                        netPotential += 4 * EPS_STAR * (sor12 - sor6);
                        for (int k = 0; k < 3; k++) {
                            gpuAtom[atomi].accelerations[k] += (forceOverR * distArr[k] / MASS);
                            gpuAtom[atomj].accelerations[k] -= (forceOverR * distArr[k] / MASS);
                        }
                    }
                }
            }
        }
    }
}

__host__
double calcForces(thrust::host_vector<Atom> &atomList, int (&cellInteractionIndexes)[343][14]) { // Cell pairs method to calculate forces

    double netPotential = 0;
    int c; // Indexes of cell coordinates
    std::array<int, 3> cell; // Array to keep track of coordinates of a cell
    int numCellsYZ = numCellsPerDirection * numCellsPerDirection;
    int numCellsXYZ = numCellsPerDirection * numCellsPerDirection * numCellsPerDirection;
    thrust::device_vector<int> atomsInCells;
    atomsInCells.reserve(numCellsXYZ);
    thrust::device_vector<Atom> gpuAtoms = atomList;

    for (int j = 0; j < N; j++) { // Set all accelerations equal to zero
        for (int i = 0; i < 3; ++i) {
            atomList[j].accelerations[i] = 0;
        }
    }

    for (int i = 0; i < N; i++) { // Place atoms in cells
        for (int j = 0; j < 3; j++) {
            cell[j] = atomList[i].positions[j] / cellLength; // Find the coordinates of a cell an atom belongs to
        }
        // Turn coordinates of cell into a cell index for the header array
        c = cell[0] * numCellsYZ + cell[1] * numCellsPerDirection + cell[2];
        atomsInCells[c].push_back(i);
    }

    for (int c = 0; c < numCellsXYZ; c++) {
         calcForcesOnCell<<<1, 1>>>(c, gpuAtoms, cellInteractionIndexes, atomsInCells);
    }
    hipDeviceSynchronize();
    atomList = gpuAtoms;
    return netPotential;
}

__host__
std::vector<Atom> simpleCubicCell() {
    double n = std::cbrt(N); // Number of atoms in each dimension

    std::vector<Atom> atomList;
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            for (int k = 0; k < n; k++) {
                atomList.push_back(Atom(i * SIGMA, j * SIGMA, k * SIGMA));
            }
        }
    }
    return atomList;
}

__host__
thrust::host_vector<Atom> faceCenteredCell() {
    // Each face centered unit cell has four atoms
    // Method creates a cubic arrangement of face centered unit cells

    double n = std::cbrt(N / 4.0); // Number of unit cells in each direction
    double dr = L / n; // Distance between two corners in a unit cell
    double dro2 = dr / 2.0; // dr over 2

    thrust::host_vector<Atom> atomList;

    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            for (int k = 0; k < n; k++) {
                atomList.push_back(Atom(i * dr, j * dr, k * dr));
                atomList.push_back(Atom(i * dr + dro2, j * dr + dro2, k * dr));
                atomList.push_back(Atom(i * dr + dro2, j * dr, k * dr + dro2));
                atomList.push_back(Atom(i * dr, j * dr + dro2, k * dr + dro2));
            }
        }
    }
    return atomList;
}


__host__
void radialDistribution() {
    
    std::string line;
    std::string s;

    int numDataPts = 100;
    double data[numDataPts];
    std::array<double, N> x;
    std::array<double, N> y;
    std::array<double, N> z;
    // Arrays hold coordinates of each atom at each step
    double dr = L / 2.0 / 100;

    std::ifstream xyz ("out.xyz");

    for (int i = 0; i < numTimeSteps; i++) {

        std::getline(xyz, line); // Skips line with number of molecules
        std::getline(xyz, line); // Skips comment line

        for (int row = 0; row < N; row++) {
            std::getline(xyz, line);
            std::istringstream iss( line );

            iss >> s >> x[row] >> y[row] >> z[row]; // Drop atom type, store coordinates of each atom
        }
        

        if (i >= numTimeSteps / 2) {
            for (int j = 0; j < N - 1; j++) {
                for (int k = j + 1; k < N; k++) {
                    double xDif = x[j] - x[k]; // Distance between atoms in x direction
                    xDif = xDif - L * std::round(xDif / L); // Boundary conditions
                    double yDif = y[j] - y[k];
                    yDif = yDif - L * std::round(yDif / L);
                    double zDif = z[j] - z[k];
                    zDif = zDif - L * std::round(zDif / L);
                    
                    double r = std::sqrt(dot(xDif, yDif, zDif));

                    if (r < L/2.0) {
                        data[(int)(r / dr)] += 2.0;
                    }
                }
            }
        }
    }
    xyz.close();
    std::ofstream radialData("Radial_Data.dat");

    radialData << "r \t \t g(r) \n";
    for (int i = 0; i < numDataPts; i++) {
        double r = (i + .5) * dr;
        data[i] /= (numTimeSteps / 2.0);
        data[i] /= 4.0 * M_PI / 3.0 * (std::pow(i + 1, 3.0) - std::pow(i, 3.0)) * std::pow(dr, 3.0) * rho;
        radialData << r << " , " << data[i] / N << "\n";
    }
    radialData.close();
}
