#include "hip/hip_runtime.h"
/*
Host: CPU
Device: GPU
    __global__ - Runs on the GPU, called from the CPU or the GPU*. Executed with <<<dim3>>> arguments.
    __device__ - Runs on the GPU, called from the GPU. Can be used with variabiles too.
    __host__ - Runs on the CPU, called from the CPU.
     __global__ functions can be called from other __global__ functions starting compute capability 3.5.
*/
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

#include "cuPrintf.cu"
#include "cuPrintf.cuh"
#include <hip/hip_runtime.h>

#include <iostream>
#include <sstream>
#include <cmath>
#include <random>
#include <fstream>
#include <array>
#include <string>

struct Atom {
    double positions[3];
    double velocities[3] = {0,0,0};
    double accelerations[3] = {0,0,0};
    double oldAccelerations[3] = {0,0,0};

    Atom(double x, double y, double z) {
        positions[0] = x;
        positions[1] = y;
        positions[2] = z;
    }
    Atom() {

    }
};

const double Kb = 1.38064582 * std::pow(10, -23); // J / K
const double Na = 6.022 * std::pow(10, 23); // Atoms per mole

const int numTimeSteps = 5000; // Parameters to change for simulation
const double dt_star= .001;

const int N = 32; // Number of atoms in simulation
const double SIGMA = 3.405; // Angstroms
const double EPSILON = 1.6540 * std::pow(10, -21); // Joules
const double EPS_STAR = EPSILON / Kb; // ~ 119.8 K

const double rhostar = .45; // Dimensionless density of gas
const double rho = rhostar / std::pow(SIGMA, 3); // Density of gas
const double L = std::cbrt(N / rho); // Unit cell length
const double rCutoff = SIGMA * 2.5; // Forces are negligible past this distance, not worth calculating
const double rCutoffSquared = rCutoff * rCutoff;
const double tStar = 1.24; // Reduced units of temperature
const double TARGET_TEMP = tStar * EPS_STAR;
// 39.9 is mass of argon in amu, 10 is a conversion between the missing units :)
const double MASS = 39.9 * 10 / Na / Kb; // Kelvin * ps^2 / A^2
const double timeStep = dt_star * std::sqrt(MASS * SIGMA * SIGMA / EPS_STAR); // Convert time step to picoseconds

// Declarations
void thermostat(Atom *atomList);
double calcForces(Atom *atomList);
thrust::host_vector<Atom> faceCenteredCell();
std::vector<Atom> simpleCubicCell();
void radialDistribution();



__host__
void writePositions(Atom *atomList, std::ofstream &positionFile, int i) {
    positionFile << N << "\nTime: " << i << "\n";
    for (int j = 0; j < N; ++j) { // Write positions to xyz file
        positionFile << "A " << atomList[j].positions[0] << " " << atomList[j].positions[1] << " " << atomList[j].positions[2] << "\n";
    }
}

int main() {

    std::ofstream positionFile("out.xyz");
    //std::ofstream debug("debug.dat");

    // Arrays to hold energy values at each step of the process
    std::vector<double> KE;
    std::vector<double> PE;
    std::vector<double> netE;

    std::random_device rd;
    std::default_random_engine generator(3); // (rd())
    std::uniform_real_distribution<double> distribution(-1.0, 1.0);

    thrust::host_vector<Atom> atomList = faceCenteredCell();
    Atom atoms[N];
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < 3; j++) {
            atoms[i].positions[j] = atomList[i].positions[j];
            atoms[i].velocities[j] = atomList[j].velocities[j];
        }
    }

    for (int i = 0; i < N; ++i) { // Randomize velocities
         for (int j = 0; j < 3; ++j) {
             atoms[i].velocities[j] = distribution(generator);
         }
    }
   
    thermostat(atoms); // Make velocities more accurate

    double totalVelSquared;
    double netPotential;

    double count = .01;
    for (int i = 0; i < numTimeSteps; ++i) { // Main loop handles integration and printing to files

        if (i > count * numTimeSteps) { // Percent progress
            std::cout << count * 100 << "% \n";
            count += .01;
        }

        writePositions(atoms, positionFile, i);

        for (int k = 0; k < N; ++k) { // Update positions
            for (int j = 0; j < 3; ++j) {
                atoms[k].positions[j] += atoms[k].velocities[j] * timeStep 
                    + .5 * atoms[k].accelerations[j] * timeStep * timeStep;
                atoms[k].positions[j] += -L * std::floor(atoms[k].positions[j] / L); // Keep atom inside box
                atoms[k].oldAccelerations[j] = atomList[k].accelerations[j];
            }
        }

        netPotential = calcForces(atoms); // Update accelerations and return potential of system

        totalVelSquared = 0;
        for (int k = 0; k < N; ++k) { // Update velocities
            for (int j = 0; j < 3; ++j) {
                atoms[k].velocities[j] += .5 * (atoms[k].accelerations[j] + atoms[k].oldAccelerations[j]) * timeStep;
                totalVelSquared += atoms[k].velocities[j] * atoms[k].velocities[j];
            }
        }

        if (i < numTimeSteps / 2 && i % 5 == 0) { // Apply velocity modifications for first half of sample
            thermostat(atoms);
        }

        if (i > numTimeSteps / 2) { // Record energies after half of time has passed
            double netKE = .5 * MASS * totalVelSquared;
            KE.push_back(netKE);
            PE.push_back(netPotential);
            netE.push_back(netPotential + netKE);
        }
    }

    double avgPE = 0; // Average PE array
    for (double i : PE) {
        avgPE += i;
    }
    avgPE /= PE.size();
    std::cout << "Avg PE: " << avgPE << std::endl;

    double SoLo2 = SIGMA / (L / 2); // Sigma over L over 2
    double Ulrc = (8.0 / 3.0) * M_PI * N * rhostar * EPS_STAR; // Potential sub lrc (long range corrections)
    double temp = 1.0 / 3.0 * std::pow(SoLo2, 9.0);
    double temp1 = std::pow(SoLo2, 3.0);
    Ulrc *= (temp - temp1);
    double PEstar = ((avgPE + Ulrc) / N) / EPS_STAR; // Reduced potential energy

    std::cout << "Reduced potential with long range correction: " << PEstar << std::endl;

    positionFile.close();
    //debug.close();

    // std::cout << "Finding radial distribution \n";
    // radialDistribution(); // Comment out function to reduce runtime

    return 0;
}

__device__
void dotForGPU(double x, double y, double z, double &r2) { // Returns dot product of a vector
    r2 = x * x + y * y + z * z;
}

__host__
double dotForCPU(double x, double y, double z) {
    return x * x + y * y + z * z;
}

__host__
void thermostat(Atom *atomList) {
    double instantTemp = 0;
    for (int i = 0; i < N; i++) { // Add kinetic energy of each molecule to the temperature
        instantTemp += MASS * dotForCPU(atomList[i].velocities[0], atomList[i].velocities[1], atomList[i].velocities[2]);
    }
    instantTemp /= (3 * N - 3);
    double tempScalar = std::sqrt(TARGET_TEMP / instantTemp);
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < 3; ++j) {
            atomList[i].velocities[j] *= tempScalar; // V = V * lambda
        }
    }
}

__global__
void calcForcesPerAtom(Atom *devAtoms, int atomidx, double L, double EPS_STAR, double MASS, double *netPotential) {
    double distArr[3]; // Record distance between atoms
    double localPotential = 0;
    double r2;

    for(int j = 0; j < N; j++) {
        for (int k = 0; k < 3; k++) {
            // Apply boundary conditions
            distArr[k] = devAtoms[atomidx].positions[k] - devAtoms[j].positions[k];
            distArr[k] -= L * std::round(distArr[k] / L);
        }
        r2 = distArr[0] * distArr[0] + distArr[1] * distArr[1] + distArr[2] * distArr[2]; // Dot product b/t atoms
        if (r2 < rCutoffSquared) {
            double s2or2 = SIGMA * SIGMA / r2; // Sigma squared over r squared
            double sor6 = s2or2 * s2or2 * s2or2; // Sigma over r to the sixth
            double sor12 = sor6 * sor6; // Sigma over r to the twelfth

            double forceOverR = 24 * EPS_STAR / r2 * (2 * sor12 - sor6);
            localPotential += 4 * EPS_STAR * (sor12 - sor6);
            for (int k = 0; k < 3; k++) {
                devAtoms[atomidx].accelerations[k] += (forceOverR * distArr[k] / MASS);
                devAtoms[j].accelerations[k] -= (forceOverR * distArr[k] / MASS);
            }
        }
    }
    *netPotential += localPotential;
}

__host__
double calcForces(Atom *atomList) { // Cell pairs method to calculate forces
    double *netPotential;
    hipMallocManaged(&netPotential, sizeof(double));
    *netPotential = 0;

    for (int j = 0; j < N; j++) { // Set all accelerations equal to zero
        for (int i = 0; i < 3; ++i) {
            atomList[j].accelerations[i] = 0;
        }
    }
    Atom *devAtoms;
    hipMallocManaged(&devAtoms, N * sizeof(Atom));
    hipMemcpy(devAtoms, atomList, N * sizeof(Atom), hipMemcpyHostToDevice);

    for (int c = 0; c < N; c++) {
         calcForcesPerAtom<<<1, 1>>>(devAtoms, c, L, EPS_STAR, MASS, netPotential);
    }
    hipDeviceSynchronize();
    hipMemcpy(atomList, devAtoms, N * sizeof(Atom), hipMemcpyDeviceToHost);
    hipFree(devAtoms);
    double result = *netPotential;
    hipFree(netPotential);
    return result;
}

__host__
std::vector<Atom> simpleCubicCell() {
    double n = std::cbrt(N); // Number of atoms in each dimension

    std::vector<Atom> atomList;
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            for (int k = 0; k < n; k++) {
                atomList.push_back(Atom(i * SIGMA, j * SIGMA, k * SIGMA));
            }
        }
    }
    return atomList;
}

__host__
thrust::host_vector<Atom> faceCenteredCell() {
    // Each face centered unit cell has four atoms
    // Method creates a cubic arrangement of face centered unit cells

    double n = std::cbrt(N / 4.0); // Number of unit cells in each direction
    double dr = L / n; // Distance between two corners in a unit cell
    double dro2 = dr / 2.0; // dr over 2

    thrust::host_vector<Atom> atomList;

    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            for (int k = 0; k < n; k++) {
                atomList.push_back(Atom(i * dr, j * dr, k * dr));
                atomList.push_back(Atom(i * dr + dro2, j * dr + dro2, k * dr));
                atomList.push_back(Atom(i * dr + dro2, j * dr, k * dr + dro2));
                atomList.push_back(Atom(i * dr, j * dr + dro2, k * dr + dro2));
            }
        }
    }
    return atomList;
}


__host__
void radialDistribution() {
    
    std::string line;
    std::string s;

    int numDataPts = 100;
    double data[numDataPts];
    std::array<double, N> x;
    std::array<double, N> y;
    std::array<double, N> z;
    // Arrays hold coordinates of each atom at each step
    double dr = L / 2.0 / 100;

    std::ifstream xyz ("out.xyz");

    for (int i = 0; i < numTimeSteps; i++) {

        std::getline(xyz, line); // Skips line with number of molecules
        std::getline(xyz, line); // Skips comment line

        for (int row = 0; row < N; row++) {
            std::getline(xyz, line);
            std::istringstream iss( line );

            iss >> s >> x[row] >> y[row] >> z[row]; // Drop atom type, store coordinates of each atom
        }
        

        if (i >= numTimeSteps / 2) {
            for (int j = 0; j < N - 1; j++) {
                for (int k = j + 1; k < N; k++) {
                    double xDif = x[j] - x[k]; // Distance between atoms in x direction
                    xDif = xDif - L * std::round(xDif / L); // Boundary conditions
                    double yDif = y[j] - y[k];
                    yDif = yDif - L * std::round(yDif / L);
                    double zDif = z[j] - z[k];
                    zDif = zDif - L * std::round(zDif / L);
                    
                    double r = std::sqrt(dotForCPU(xDif, yDif, zDif));

                    if (r < L/2.0) {
                        data[(int)(r / dr)] += 2.0;
                    }
                }
            }
        }
    }
    xyz.close();
    std::ofstream radialData("Radial_Data.dat");

    radialData << "r \t \t g(r) \n";
    for (int i = 0; i < numDataPts; i++) {
        double r = (i + .5) * dr;
        data[i] /= (numTimeSteps / 2.0);
        data[i] /= 4.0 * M_PI / 3.0 * (std::pow(i + 1, 3.0) - std::pow(i, 3.0)) * std::pow(dr, 3.0) * rho;
        radialData << r << " , " << data[i] / N << "\n";
    }
    radialData.close();
}
